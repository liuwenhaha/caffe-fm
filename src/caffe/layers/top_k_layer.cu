#include "hip/hip_runtime.h"
#include "caffe/layers/top_k_layer.hpp"
#include <cstdio>
#include <algorithm>


namespace caffe {

using std::sort;

template <typename Dtype>
__global__ void TopKForward(const int nthreads, const Dtype* bottom_data,
    const int* ids, const int chw, Dtype* top_data) {

  CUDA_KERNEL_LOOP(index, nthreads) {
    int ind = index % chw;
    int b_ind = index / chw;
    int bottom_index = ids[b_ind] * chw + ind;
    top_data[index] = bottom_data[bottom_index];
  }
}

template <typename Dtype>
void TopKLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  // top batch size
  k_ = bottom[2]->shape(0);
  // score data
  sort_by_score.score_ = bottom[1]->mutable_cpu_data();
  // buffer
  ids_.Reshape(bottom[0]->shape(0), 1, 1, 1);
  for (int i = 0; i < bottom[0]->shape(0); ++i)
    ids_.mutable_cpu_data()[i] = i;
  int *ids_cpu_data = ids_.mutable_cpu_data();
  // sort by score
  sort(ids_cpu_data, ids_cpu_data + bottom[0]->shape(0), sort_by_score);
  // choose first k_
  sort(ids_cpu_data, ids_cpu_data + k_);

  // copy
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int* ids_gpu_data = ids_.gpu_data();
  const int count = k_ * channels_ * height_ * width_;
  TopKForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>> (
      count, bottom_data, ids_gpu_data, channels_ * height_ * width_, top_data);
  CUDA_POST_KERNEL_CHECK;
  if (top.size() == 2) {
    for (int i = 0; i < top[1]->shape(0); ++i)
      top[1]->mutable_cpu_data()[i] = ids_cpu_data[i];
  }
}

template <typename Dtype>
__global__ void TopKBackward(const int nthreads, const Dtype* top_diff,
    const int *ids, const int chw, Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int ind = index % chw;
    int b_ind = index / chw;
    int bottom_index = ids[b_ind] * chw + ind;
    bottom_diff[bottom_index] = top_diff[index];
  }
}

template <typename Dtype>
void TopKLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0])
    return;
  const Dtype* top_diff = top[0]->gpu_diff();
  const int* ids_gpu_data = ids_.gpu_data();
  Dtype *bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = k_ * channels_ * height_ * width_;
  caffe_gpu_set(bottom[0]->shape(0) * bottom[0]->shape(1) * bottom[0]->shape(2) * bottom[0]->shape(3), Dtype(0.), bottom_diff);
  TopKBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, top_diff, ids_gpu_data, channels_ * height_ * width_, bottom_diff);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(TopKLayer);


}
